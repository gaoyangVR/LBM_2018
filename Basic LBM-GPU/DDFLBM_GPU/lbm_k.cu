#include <hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<hip/hip_vector_types.h>
#include "utility.h"

__constant__ int NX;
__constant__ int NY;
__constant__ int NZ;
__constant__ int Qm;
__constant__ int gnum;



void copyConstantstoGPU(int nx, int ny, int nz,int qm,int num)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(NX), &nx, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(NY), &ny, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(NZ), &nz, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Qm), &qm, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gnum), &num, sizeof(int)));
	
	}



